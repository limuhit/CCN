#include "hip/hip_runtime.h"
#include "d_input.hpp"
#include <hiprand.h>
#include <stdio.h>
#include <math.h>
#include <float.h>
#include "math_functions.hpp"

void d_input_opt::init(){
    init_base();
}

void d_input_opt::reshape(int num, int channel, int height, int width){
    if (!reshape_base(num, channel, height, width)) return; 
    pidx_ = 0;
    mod_ = height_ + width_ + channel_ - 2;
    index_ = at::zeros({height_,width_},at::kInt);
    int * idx = index_.data_ptr<int>();
    start_idx_.clear();
    int index = 0;
    for (int ps = 0; ps < height_ + width_ - 1; ps++) {
        start_idx_.push_back(index);
        for (int i = 0; i < height_; i++) {
            int j = ps - i;
            if (j < 0 || j >= width_)
                continue;
            idx[index] = i*width_ + j;
            index++;
        }
    }
    start_idx_.push_back(index);
    index_ = index_.to(torch::Device(torch::kCUDA, device_));
}

void d_input_opt::reshape_top(at::TensorOptions option){
    std::vector<std::vector<int64_t>> shapes;
    shapes.push_back({num_,channel_,height_,width_});
    reshape_top_base(option,shapes);
}


template <typename scalar_t>
__global__ void d_input_forward_kernel(const int num, const scalar_t * const input,
    const int * index, scalar_t * const output, const int start_idx, const int len_idx,
    const int height, const int width, const int nchannel, const int psum) {
    CUDA_KERNEL_LOOP(i, num) {

        int tl = i  % len_idx;
        int tn = i / len_idx;
        int thw = index[tl + start_idx];
        int tw = thw % width;
        int th = thw / width;
        int tc = psum - tw - th;
        int pidx = (tn*nchannel + tc)*height*width + thw;
        output[pidx] = input[i];
    }

}


std::vector<at::Tensor>  d_input_opt::forward_cuda(at::Tensor  bottom_data) 
{
    reshape(bottom_data.size(0), channel_, bottom_data.size(2), bottom_data.size(3));
    reshape_top(bottom_data.options());
    const int* index = index_.data_ptr<int>();
    int psum = pidx_;
    pidx_ = (pidx_ + 1) % mod_;
	AT_DISPATCH_FLOATING_TYPES(
		bottom_data.scalar_type(), "d_input_forward_cuda", 
			([&] {
                    timer_->start();
                    if (psum == 0) {
                        caffe_gpu_set(stream_, num_*channel_*width_*height_, scalar_t(0), top_data_[0].data_ptr<scalar_t>());
                    }
                    else {
                        psum -= 1;
                        int st = psum - channel_ + 1 < 0 ? 0 : psum - channel_ + 1;
                        int end = psum < height_ + width_ - 2 ? psum + 1 : height_ + width_ - 1;
                        int len_idx = start_idx_[end] - start_idx_[st];
                        int count = len_idx*num_ * 1;
                        d_input_forward_kernel << <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS, 0, stream_ >> >
                            (count, bottom_data.data_ptr<scalar_t>(), index, top_data_[0].data_ptr<scalar_t>(), 
                                start_idx_[st], len_idx, height_, width_,  channel_, psum);
                    }
                    CUDA_POST_KERNEL_CHECK;
                    timer_->stop("kernel 1");
   			    }
			)
    );
    return top_data_;
}

