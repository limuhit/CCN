#include "hip/hip_runtime.h"
#include "dtow.hpp"
#include <hiprand.h>
#include <stdio.h>
#include <math.h>
#include <float.h>

void dtow_opt::init(){
   init_base();
}

void dtow_opt::reshape(int num, int channel, int height, int width){
    if (!reshape_base(num, channel, height, width)) return; 
    if(d2w_){
        h_out_ = height_ * stride_;
        w_out_ = width_ * stride_;
        ch_out_ = channel_ / stride_ / stride_;
    }else{
        h_out_ = height_ / stride_;
        w_out_ = width_ / stride_;
        ch_out_ = channel_ * stride_ * stride_;
    }
}

void dtow_opt::reshape_top(at::TensorOptions options){
    std::vector<std::vector<int64_t>> shapes;
    shapes.push_back({num_,ch_out_, h_out_, w_out_});
    reshape_top_base(options,shapes);
}

void dtow_opt::reshape_bottom(at::TensorOptions options){
    std::vector<std::vector<int64_t>> shapes;
    shapes.push_back({num_,channel_,height_,width_});
    reshape_bottom_base(options,shapes);
}


template <typename scalar_t>
__global__ void dtow_forward_kernel(const int nthreads, const scalar_t* const bottom_data,
    const int num, const int channels, const int height, const int width,
    const int channels_out, const int height_out, const int width_out, const int patch_size,
    scalar_t* const top_data) {
    CUDA_KERNEL_LOOP(index, nthreads) {
        int tw = index%width;
        int th = (index / width) % height;
        int tc = (index / width / height) % channels;
        int	tn = index / width / height / channels;
        int p2size = patch_size*patch_size;
        int pc = tc / p2size;
        int rc = tc % p2size;
        int ph = th*patch_size + rc / patch_size;
        int pw = tw*patch_size + rc % patch_size;
        int pidx = ((tn*channels_out + pc)*height_out + ph)*width_out + pw;
        top_data[pidx] = bottom_data[index];
    }
}

template <typename scalar_t>
__global__ void wtod_forward_kernel(const int nthreads, const scalar_t* const bottom_data,
    const int num, const int channels, const int height, const int width,
    const int channels_out, const int height_out, const int width_out, const int patch_size,
    scalar_t* const top_data) {
    CUDA_KERNEL_LOOP(index, nthreads) {
        int tw = index%width;
        int th = (index / width) % height;
        int tc = (index / width / height) % channels;
        int	tn = index / width / height / channels;
        int p2size = patch_size*patch_size;
        int ph = th / patch_size;
        int pw = tw / patch_size;
        int pc = tc * p2size + (th%patch_size)*patch_size + tw%patch_size;
        int pidx = ((tn*channels_out + pc)*height_out + ph)*width_out + pw;
        top_data[pidx] = bottom_data[index];
    }
}


std::vector<at::Tensor>  dtow_opt::forward_cuda(at::Tensor  bottom_data) 
{
    reshape(bottom_data.size(0), bottom_data.size(1), bottom_data.size(2), bottom_data.size(3));
    reshape_top({bottom_data.options()});
	int count;
	AT_DISPATCH_FLOATING_TYPES(
		bottom_data.scalar_type(), "dtow_forward_cuda", 
			([&] {
                    timer_->start();
                    count = num_ * channel_ * width_ * height_;
                    if(d2w_){
                        dtow_forward_kernel<< <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS, 0 ,stream_ >> >
                            (count, bottom_data.data_ptr<scalar_t>(), num_, channel_, height_, width_, 
                                ch_out_, h_out_, w_out_, stride_, top_data_[0].data_ptr<scalar_t>());
                    }else{
                        wtod_forward_kernel<< <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS, 0 ,stream_ >> >
                            (count, bottom_data.data_ptr<scalar_t>(), num_, channel_, height_, width_, 
                                ch_out_, h_out_, w_out_, stride_, top_data_[0].data_ptr<scalar_t>());
                    }
                    CUDA_POST_KERNEL_CHECK;
                    timer_->stop("kernel 1");
   			    }
			)
    );
    return top_data_;
}

template <typename scalar_t>
__global__ void dtow_backward_kernel(const int nthreads, const scalar_t* const top_diff,
    const int num, const int channels, const int height, const int width,
    const int channels_out, const int height_out, const int width_out, const int patch_size,
    scalar_t* const bottom_diff) {
    CUDA_KERNEL_LOOP(index, nthreads) {
        int tw = index%width;
        int th = (index / width) % height;
        int tc = (index / width / height) % channels;
        int	tn = index / width / height / channels;
        int p2size = patch_size*patch_size;
        int pc = tc / p2size;
        int rc = tc % p2size;
        int ph = th*patch_size + rc / patch_size;
        int pw = tw*patch_size + rc % patch_size;
        int pidx = ((tn*channels_out + pc)*height_out + ph)*width_out + pw;
        bottom_diff[index] = top_diff[pidx];
    }
}

template <typename scalar_t>
__global__ void wtod_backward_kernel(const int nthreads, const scalar_t* const top_diff,
    const int num, const int channels, const int height, const int width,
    const int channels_out, const int height_out, const int width_out, const int patch_size,
    scalar_t* const bottom_diff) {
    CUDA_KERNEL_LOOP(index, nthreads) {
        int tw = index%width;
        int th = (index / width) % height;
        int tc = (index / width / height) % channels;
        int	tn = index / width / height / channels;
        int p2size = patch_size*patch_size;
        int ph = th / patch_size;
        int pw = tw / patch_size;
        int pc = tc * p2size + (th%patch_size)*patch_size + tw%patch_size;
        int pidx = ((tn*channels_out + pc)*height_out + ph)*width_out + pw;
        bottom_diff[index] = top_diff[pidx];
    }
}

std::vector<at::Tensor>  dtow_opt::backward_cuda(at::Tensor  top_diff) 
{
    reshape_bottom({top_diff.options()});
	int count;
	AT_DISPATCH_FLOATING_TYPES(
		top_diff.scalar_type(), "dtow_backward_cuda", 
			([&] {
                    timer_->start();
                    count = num_ * channel_ * width_ * height_;
                    if(d2w_){
                        dtow_backward_kernel<< <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS, 0 ,stream_ >> >
                            (count, top_diff.data_ptr<scalar_t>(), num_, channel_, height_, width_, 
                                ch_out_, h_out_, w_out_, stride_, bottom_diff_[0].data_ptr<scalar_t>());
                    }else{
                        wtod_backward_kernel<< <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS, 0 ,stream_ >> >
                            (count, top_diff.data_ptr<scalar_t>(), num_, channel_, height_, width_, 
                                ch_out_, h_out_, w_out_, stride_, bottom_diff_[0].data_ptr<scalar_t>());
                    }
                    
                    CUDA_POST_KERNEL_CHECK;
                    timer_->stop("kernel 1");
   			    }
			)
    );
    return bottom_diff_;
}